
#include <hip/hip_runtime.h>
__global__ void loop_unrolling(
    bool *arr,
    unsigned int *shape,
    unsigned long long shape_total,
    unsigned long long shape_count,
    unsigned long long step,
    unsigned char order
)
{
    unsigned long long idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long idx_full;
    unsigned int i = 0;
    unsigned int *indices = new unsigned int[shape_count]; // array to hold the computed indices
    unsigned long long tmp;
    
    idx_full = i * step + idx;
    while (idx_full < shape_total)
    {
        tmp = idx_full;
        for (unsigned int j = 0; j < shape_count; ++j)
        {
            unsigned int dimension = (order == 0) ? shape_count - j - 1 : j;
            indices[dimension] = tmp % shape[dimension];
            tmp /= shape[dimension];
        }
        // Set true if any index equals to 1
        for (unsigned int j = 0; j < shape_count; ++j)
        {
            if (indices[j] == 1)
            {
                arr[idx_full] = true;
                break;
            }
            arr[idx_full] = false;
        }
        i += 1;
        idx_full = i * step + idx;
    }
    delete[] indices;
}
