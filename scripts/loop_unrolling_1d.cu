
#include <hip/hip_runtime.h>
__global__ void loop_unrolling(
    bool *arr,
    unsigned long long shape,
    unsigned long long step
)
{
    unsigned char idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned char i = 0;
    
    while (i * step + idx < shape) {
        arr[i * step + idx] = !arr[i * step + idx];
        i += 1;
    }
}