
#include <hip/hip_runtime.h>
__global__ void loop_unrolling(
    bool *arr,
    unsigned int *shape,
    unsigned long long gpu_arr_size,
    unsigned long long shape_total,
    unsigned long long dimensions_count,
    unsigned long long step,
    unsigned char order,
    unsigned long long batch_start
)
{
    unsigned long long idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long idx_full;
    unsigned int i = 0;
    unsigned int *indices = new unsigned int[dimensions_count]; // array to hold the computed indices
    unsigned long long tmp;
    
    idx_full = i * step + idx;

    while (idx_full < shape_total && idx_full < gpu_arr_size)
    {
        //tmp = idx_full;
        tmp = idx_full + batch_start; // add batch_start to account for the offset
        // Compute the indices
        for (unsigned int j = 0; j < dimensions_count; ++j)
        {
            unsigned int dimension = (order == 0) ? dimensions_count - j - 1 : j;
            // Modulo by the dimension size
            indices[dimension] = tmp % shape[dimension];
            // Divide by the dimension size
            tmp /= shape[dimension];
        }
        printf("idx_full: %llu, idx: %llu, batch_start: %llu\n", idx_full, idx, batch_start);
        // Set true if any index equals to 1
        for (unsigned int j = 0; j < dimensions_count; ++j)
        {
            // j is the dimension
            if (indices[j] == 1)
            {
                arr[idx_full] = true;
                break;
            }
        }
        i += 1;
        idx_full = i * step + idx;
    }
    // Free the memory
    delete[] indices;
}
